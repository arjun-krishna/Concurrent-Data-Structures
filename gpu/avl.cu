#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

typedef struct node {
	int data;
	struct node *parent;
	struct node *left;
	struct node *right;
	int sema;
	int height;
} node;

/*
__device__ int max(int a, int b)
{
    if(a > b)
			return a;
		return b;
}
*/

__device__ int lock(node* n) {
	return !atomicExch(&n->sema, 1);
}

__device__ void unlock(node* n) {
	atomicExch(&n->sema, 0);
}

__device__ node* new_node(int val, node* parent) {
	node *tmp = (node *) malloc(sizeof(node));
	tmp->data = val;
	tmp->parent = parent;
	tmp->left = tmp->right = NULL;
	tmp->height = 1;
	return tmp;
}

__device__ int height(node *root)
{
    if (root == NULL)
        return 0;
    return root->height;
}

__device__ node* left_rotate(node* root)
{
    node* temp1 = root->right;
    node* temp2 = temp1->left;
 
    temp1->left = root;
    root->right = temp2;
 
    root->height = max(height(root->left), height(root->right))+1;
    temp1->height = max(height(temp1->left), height(temp1->right))+1;
 
    return temp1;
}

__device__ node* right_rotate(node* root)
{
    node* temp1 = root->left;
    node* temp2 = temp1->right;
 
    temp1->right = root;
    root->left = temp2;
 
    root->height = max(height(root->left), height(root->right))+1;
    temp1->height = max(height(temp1->left), height(temp1->right))+1;
 
    return temp1;
}

__device__ int get_balance(node *root)
{
    if (root == NULL)
        return 0;
    return height(root->left) - height(root->right);
}

__device__ int MASTER_LOCK = 0;

__device__ void rebalance(node* p, int key) {
	bool flag = true;
	if (p->parent) {
		while (atomicExch(&(p->parent->sema), 1) && flag) {
			// acquired 
			p->height = max(height(p->left), height(p->right)) + 1;
			int balance = get_balance(p);

			if (balance > 1 && key < p->left->data) {
				if (p->data < p->parent->data) {
					p->parent->left = right_rotate(p);
				} else {
					p->parent->right = right_rotate(p);
				}
			}

			// Right Right Case
		  if (balance < -1 && key > p->right->data) {
		  	if (p->data < p->parent->data) {
		  		p->parent->left = left_rotate(p);
		  	} else {
		  		p->parent->right = left_rotate(p);
		  	}
		  }

			// Left Right Case
		  if (balance > 1 && key > p->left->data)
		  {
		  	p->left =  left_rotate(p->left);

		  	if (p->data < p->parent->data) {
					p->parent->left = right_rotate(p);
				} else {
					p->parent->right = right_rotate(p);
				}
		  }

			// Right Left Case
		  if (balance < -1 && key < p->right->data)
		  {
				p->right = right_rotate(p->right);
				
				if (p->data < p->parent->data) {
		  		p->parent->left = left_rotate(p);
		  	} else {
		  		p->parent->right = left_rotate(p);
		  	}
		  }


			atomicExch(&(p->parent->sema), 0);
			flag = false;
			rebalance(p->parent, key);
		}
	} else {																						// ROOT balance
		while(atomicExch(&MASTER_LOCK, 1) && flag) {

			p->height = max(height(p->left), height(p->right)) + 1;
			int balance = get_balance(p);

			if (balance > 1 && key < p->left->data) {
				if (p->data < p->parent->data) {
					p->parent->left = right_rotate(p);
				} else {
					p->parent->right = right_rotate(p);
				}
			}

			// Right Right Case
		  if (balance < -1 && key > p->right->data) {
		  	if (p->data < p->parent->data) {
		  		p->parent->left = left_rotate(p);
		  	} else {
		  		p->parent->right = left_rotate(p);
		  	}
		  }

			// Left Right Case
		  if (balance > 1 && key > p->left->data)
		  {
		  	p->left =  left_rotate(p->left);

		  	if (p->data < p->parent->data) {
					p->parent->left = right_rotate(p);
				} else {
					p->parent->right = right_rotate(p);
				}
		  }

			// Right Left Case
		  if (balance < -1 && key < p->right->data)
		  {
				p->right = right_rotate(p->right);
				
				if (p->data < p->parent->data) {
		  		p->parent->left = left_rotate(p);
		  	} else {
		  		p->parent->right = left_rotate(p);
		  	}
		  }

			atomicExch(&MASTER_LOCK, 0);
			flag = false;
		}
	}
}

__device__ void coarse_rebalance(node* p, int key) {
	if (p->parent) {
		p->height = max(height(p->left), height(p->right)) + 1;
		int balance = get_balance(p);

		if (balance > 1 && key < p->left->data) {
			if (p->data < p->parent->data) {
				p->parent->left = right_rotate(p);
			} else {
				p->parent->right = right_rotate(p);
			}
		}

		// Right Right Case
	  if (balance < -1 && key > p->right->data) {
	  	if (p->data < p->parent->data) {
	  		p->parent->left = left_rotate(p);
	  	} else {
	  		p->parent->right = left_rotate(p);
	  	}
	  }

		// Left Right Case
	  if (balance > 1 && key > p->left->data)
	  {
	  	p->left =  left_rotate(p->left);

	  	if (p->data < p->parent->data) {
				p->parent->left = right_rotate(p);
			} else {
				p->parent->right = right_rotate(p);
			}
	  }

		// Right Left Case
	  if (balance < -1 && key < p->right->data)
	  {
			p->right = right_rotate(p->right);
			
			if (p->data < p->parent->data) {
	  		p->parent->left = left_rotate(p);
	  	} else {
	  		p->parent->right = left_rotate(p);
	  	}
	  }

		coarse_rebalance(p->parent, key);
	}
}


__device__ void coarse_insert(node* root, int key) {

	node* curr = root;
	node* parent = NULL;
	
	if (root == NULL) {
		root = new_node(key, parent);
		return;
	}

	bool flag = true;
	while (flag) {
		if (!atomicExch(&MASTER_LOCK, 1)) {
			while (curr != NULL) {	
				parent = curr;
				if (key < curr->data)
					curr = curr->left;
				else
					curr = curr->right;	
				if (curr == NULL) {
					if (key < parent->data) {
						parent->left = new_node(key, parent);
						coarse_insert(parent->left, key);
					} else {
						parent->right = new_node(key, parent);
						coarse_insert(parent->right, key);
					}			
				} else {
					if (parent)
						atomicExch(&(parent->sema), 0);
				}
			}
			flag = false;
			atomicExch(&MASTER_LOCK, 0);
		}
	}	
}

__device__ void coarse_delete(node* root, int key) {
	return;
}





__device__ void insert(node* root, int key) {

	if (root == NULL) { 		 				// Empty Tree
		root = new_node(key, NULL); 
		return;
	}
	
	int acquired = lock(root);

	if (acquired) {
		if (key < root->data) {
			if (root->left == NULL) {			// Can be inserted to the immediate left
				root->left = new_node(key, root);
				unlock(root);
				return;
			} else {											// Release this Node and proceed
				unlock(root);
				insert(root->left, key);
			}
		} else {
			if (root->right == NULL) {		// Can be inserted to the immediate right
				root->right = new_node(key, root);
				unlock(root);
				return;
			} else {
				unlock(root);								// Release this Node and proceed
				insert(root->right, key);
			}
		}
	} else {
		insert(root, key);
	}
}

__device__ node* find(node* root, int key) {
	if (root == NULL) return NULL;

	if (root->data == key) return root;
	else if (root->data > key) return find(root->left, key);
	else return find(root->right, key);
}


__device__ void pre_order(node* root)
{
    if(root != NULL)
    {
        printf("%d ", root->data);
        pre_order(root->left);
        pre_order(root->right);
    }
		return;
}

__device__ void in_order(node* root)
{
    if(root != NULL)
    {
      in_order(root->left);
      printf("%d ", root->data);
      in_order(root->right);
    }
		return;
}

// __device__ node* min_BST(node* Node) {
// 	if (Node == NULL) return NULL;
// 	node* tmp = Node;
// 	while(tmp->left != NULL)	tmp = tmp->left;
// 	return tmp;
// }

// __device__ node* delete(node* root, int key) {
// 	if (root == NULL) return NULL;

// 	if (key < root->data) 
// 		root->left  = delete(root->left, key);
// 	else if (key > root->data) 
// 		root->right = delete(root->right, key); 
// 	else {
// 		if (root->left == NULL) {
// 			node* tmp = root->right;
// 			free(root);
// 			return tmp;
// 		} 
// 		else if (root->right == NULL) {
// 			node* tmp = root->left;
// 			free(root);
// 			return tmp;
// 		}
// 		// successor
// 		node *tmp = min_BST(root->right);
// 		root->data = tmp->data;
// 		root->right = delete(root->right, tmp->data);
// 	}

// 	if (root == NULL)
//       return root;

// 	root->height = max(height(root->left),height(root->right)) + 1;

// 	int balance = get_balance(root);

// 	// Left Left Case
//   if (balance > 1 && get_balance(root->left) >= 0)
//   	return right_rotate(root);
 
//   // Left Right Case
// 	if (balance > 1 && get_balance(root->left) < 0)
//   {
//   	root->left =  left_rotate(root->left);
//   	return right_rotate(root);
//   }
 
//   // Right Right Case
//   if (balance < -1 && get_balance(root->right) <= 0)
//   	return left_rotate(root);
 
//   // Right Left Case
//   if (balance < -1 && get_balance(root->right) > 0)
//   {
//   	root->right = right_rotate(root->right);
//   	return left_rotate(root);
//   }
 
// 	return root;
// }
