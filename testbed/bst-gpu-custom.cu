#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "../gpu/bst.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ node* root = NULL;

__global__ void initialize_kernel() {
  
  int tid = threadIdx.x;
  if (tid == 0) {
    root = new_node(0, NULL);
  }
}

__global__ void small_insert_N_kernel() {
  int tid = threadIdx.x;
  if (tid != 0) {
    insert(root, tid);
  }
}

__global__ void large_insert_kernel() {
  
  int tid = blockIdx.x*1000+threadIdx.x;
  if (tid != 0) {
    insert(root,tid);
  }
}


__global__ void insert_random() {
  int tid = threadIdx.x;
  hiprandState_t state;

  hiprand_init(tid, 0, 0, &state);

  if (tid != 0) {
    int r = hiprand(&state)%10000;
    insert(root, r);
  }
}

__global__ void small_delete_N_kernel() {
  int tid = threadIdx.x;
  if (tid%4 == 1) {
    bst_delete(root, tid);
  }
}

__global__ void large_delete_kernel() {
  
  int tid = blockIdx.x*1000+threadIdx.x;
  if (tid%4 == 1) {
    bst_delete(root, tid);
  }
}

__global__ void delete_random() {
  int tid = threadIdx.x;
  hiprandState_t state;

  hiprand_init(tid, 0, 0, &state);
  if (tid != 0) {
    int r = hiprand(&state)%10000;
    insert(root, r);
  }
}

__global__ void print_kernel() {
  // printf("In-order\n");
  in_order(root);
  printf("\n");
}


// Less than 10-threads
__global__ void custom_insert() {
  
  int tid = threadIdx.x;
  if (tid != 0) {
    switch(tid) {
      case 1 :
        insert(root, 1);
        break;
      case 2 :
        insert(root, 2);
        break;
      case 3 :
        insert(root, 3);
        break;
      case 4 :
        insert(root, 4);
        break;
      case 5 :
        insert(root, 5);
        break;
      case 6 :
        insert(root, 6);
        break;
      case 7 :
        insert(root, 7);
        break;
      case 8 :
        insert(root, 8);
        break;
      default :
        insert(root, tid);
        break;
    }
  }
}

__global__ void custom_delete() {
  int tid = threadIdx.x;
  if (tid != 0) {
    switch (tid) {
      case 1 :
        bst_delete(root, 1);
        break;
      case 2 :
        bst_delete(root, 2);
        break;
      case 3 :
        bst_delete(root, 3);
        break;
      case 4 :
        bst_delete(root, 4);
        break;
      case 5 :
        bst_delete(root, 5);
        break;
    }
  }
}

int main(int argc, char* argv[]) {
  hipDeviceSetLimit(hipLimitMallocHeapSize, 8000000); 
  
  initialize_kernel<<<1,1>>>();
  hipDeviceSynchronize();

  hipStream_t s1, s2;
  hipStreamCreate(&s1);
  hipStreamCreate(&s2);
  GPUTimer time_insert, time_delete;
  
  time_insert.Start();
  custom_insert<<<1,10, 0, s1>>>();
  time_insert.Stop();
  
  time_delete.Start();
  custom_delete<<<1,10, 0, s1>>>();
  time_delete.Stop();

  hipError_t err = hipGetLastError();  
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    return -1;
  }
  
  hipDeviceSynchronize();
  printf("Insert Kernel ran in: %f ms\n", time_insert.Elapsed());
  printf("delete Kernel ran in: %f ms\n", time_delete.Elapsed());

  print_kernel<<<1,1>>>();
  hipDeviceSynchronize();
  return 0;
}
